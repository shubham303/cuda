// cuda program to initialize array elements.

#include <stdio.h> 
#include <hip/hip_runtime.h> 
#define N 100

__global__ void arrayIntialization(int *a) {
    if (threadIdx.x < N)
        a[threadIdx.x] = threadIdx.x;
}
int main() {
    int a[N], *da; 
    int i; 
    hipMalloc( &da, N * sizeof(int)); 
    arrayIntialization<<<1,1000>>>(da);
    hipMemcpy(a, da, N * sizeof(int), hipMemcpyDeviceToHost); 
    for (i = 0; i < N; i++)
        printf("%d ", a[i]); 

    return 0; 

}