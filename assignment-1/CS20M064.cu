#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void per_row_kernel(int m, int n, int *A, int *B, int *C) {
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < m) {
		for (int i = 0; i < n; i++) {
			C[id * n + i] = A[id * n + i] + B[id * n + i];
		}

	}
}

__global__ void per_column_kernel(int m, int n, int *A, int *B, int *C) {

	int id = threadIdx.x + (blockDim.x * threadIdx.y)
			+ (blockDim.x * blockDim.y * blockIdx.x);

	if (id < n) {
		for (int i = 0; i < m; i++) {
			C[i * n + id] = A[i * n + id] + B[i * n + id];
		}
	}
}

__global__ void per_element_kernel(int m, int n, int *A, int *B, int *C) {
	int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;

	int id = (blockId * blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x)
			+ threadIdx.x;

	if (id < m * n) {
		C[id] = A[id] + B[id];
	}
}


