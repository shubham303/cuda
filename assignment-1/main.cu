#include "hip/hip_runtime.h"
#include"kernels.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <stdlib.h>

__global__ void per_row_kernel(int m, int n, int *A, int *B, int *C) {
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < m) {
		for (int i = 0; i < n; i++) {
			C[id * n + i] = A[id * n + i] + B[id * n + i];
		}

	}
}

__global__ void per_column_kernel(int m, int n, int *A, int *B, int *C) {

	int id = threadIdx.x + (blockDim.x * threadIdx.y)
			+ (blockDim.x * blockDim.y * blockIdx.x);

	if (id < n) {
		for (int i = 0; i < m; i++) {
			C[i * n + id] = A[i * n + id] + B[i * n + id];
		}
	}
}

__global__ void per_element_kernel(int m, int n, int *A, int *B, int *C) {
	int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;

	int id = (blockId * blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x)
			+ threadIdx.x;

	if (id < m * n) {
		C[id] = A[id] + B[id];
	}
}

void per_element_cpu(int m, int n, int *A, int *B, int *C) {

	for (int id = 0; id < (m * n); id++) {
		C[id] = A[id] + B[id];
	}
}

void readInput(FILE* f, int m, int n, int * A, int *B) {

	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			fscanf(f, "%d", A + (i * n + j));
		}
	}

	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			fscanf(f, "%d", B + (i * n + j));
		}
	}
}

void perRowAddition(int m, int n, int* A, int* B, int *C, int* D) {
	int threads = 0;
	int blocks = 1;
	if (m < 1024) {
		threads = m;
	} else {
		threads = 1024;
		blocks = m / 1024;
	}

	int * gpuA;
	int * gpuB;
	int * gpuC;

	hipMalloc(&gpuA, m * n * sizeof(int));
	hipMemcpy(gpuA, A, m * n * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&gpuB, m * n * sizeof(int));
	hipMemcpy(gpuB, B, m * n * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&gpuC, m * n * sizeof(int));

	time_t start, end;
	time(&start);
	per_row_kernel<<<blocks, threads>>>(m, n, gpuA, gpuB, gpuC);
	hipDeviceSynchronize();

	time(&end);

	printf("%lf \n", double(end - start));

	hipMemcpy(C, gpuC, m * n * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < m * n; i++) {
		if (C[i] != D[i]) {
			//	printf("error");
		}
	}
}

void perColumnAddition(int m, int n, int *A, int*B, int *C, int* D) {
	dim3 threads(32, 32);
	int blocks = 10;

	int * gpuA;
	int * gpuB;
	int * gpuC;

	hipMalloc(&gpuA, m * n * sizeof(int));
	hipMemcpy(gpuA, A, m * n * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&gpuB, m * n * sizeof(int));
	hipMemcpy(gpuB, B, m * n * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&gpuC, m * n * sizeof(int));

	time_t start, end;
	time(&start);
	per_column_kernel<<<blocks, threads>>>(m, n, gpuA, gpuB, gpuC);
	hipDeviceSynchronize();

	time(&end);

	printf("%lf \n", double(end - start));

	hipMemcpy(C, gpuC, m * n * sizeof(int), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	for (int i = 0; i < m * n; i++) {
		if (C[i] != D[i]) {
			//	printf("error");
		}
	}
}

void perElementAddition(int m, int n, int *A, int*B, int *C, int* D) {
	dim3 threads(32, 32);

	int x = ((m * n) / 1024) / 3;

	dim3 blocks(x, 4);

	int * gpuA;
	int * gpuB;
	int * gpuC;

	hipMalloc(&gpuA, m * n * sizeof(int));
	hipMemcpy(gpuA, A, m * n * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&gpuB, m * n * sizeof(int));
	hipMemcpy(gpuB, B, m * n * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&gpuC, m * n * sizeof(int));

	time_t start, end;
	time(&start);
	per_element_kernel<<<blocks, threads>>>(m, n, gpuA, gpuB, gpuC);
	hipDeviceSynchronize();

	time(&end);

	printf("%lf \n", double(end - start));

	hipMemcpy(C, gpuC, m * n * sizeof(int), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	for (int i = 0; i < m * n; i++) {
		if (C[i] != D[i]) {
			//printf("%d %d %d \n", C[i], D[i], i);
		}
	}
}
int main() {
	int m, n;
	int * A;
	int * B;

	int * C;
	time_t start, end;
	FILE * f = fopen("testcases/input/input3.txt", "r");
	fscanf(f, "%d %d", &m, &n);

	int *D = (int *) malloc(m * n * sizeof(int));
	FILE *w = fopen("testcases/output/output3.txt", "r");
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			fscanf(w, "%d", D + (i * n + j));
		}
	}

	m = 2;
	n = 67108864;

	A = (int *) malloc(m * n * sizeof(int));
	B = (int *) malloc(m * n * sizeof(int));
	C = (int *) malloc(m * n * sizeof(int));

	//readInput(f, m, n, A, B);

	for (int i = 0; i < m * n; i++) {
		A[i] = rand();
		B[i] = rand();
	}

	perRowAddition(m, n, A, B, C, D);
	perColumnAddition(m, n, A, B, C, D);
	perElementAddition(m, n, A, B, C, D);

	time (&start);
	per_element_cpu(m, n, A, B, C);
	time (&end);
	printf("%lf \n", double(end - start));
}
