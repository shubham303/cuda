#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
using namespace std;

struct edgepairs {
	int x;
	int y;
};

bool compareTwoEdgePairs(edgepairs a, edgepairs b) {

	if (a.y != b.y)
		return a.y < b.y;
	if (a.x != b.x)
		return a.x < b.x;

	return true;
}

// complete the following kernel...
__global__ void dkernel_Adds(int *gpuOA, int *gpuCA, int *gpulocals,
		int *gpucurrentupdate) {
	__shared__ int sum;

	/*int id = blockDim.x*blockIdx.x + threadIdx.x+1;*/

	int id = blockIdx.x + 1;
	int threadId = gridDim.y * blockIdx.y + threadIdx.x;
	int thread = threadIdx.x;
	if (thread == 0) {
		sum=0;
	}

	__syncthreads();

	int i = gpuOA[id - 1] + threadId;
	int j = gpuOA[id];
	if (i < j) {
		atomicAdd(&sum, gpucurrentupdate[gpuCA[i]]);
	}
	__syncthreads();
	if (thread == 0) {
		atomicAdd(&gpulocals[id - 1], sum);
	}

}

// complete the following kernel...
__global__ void dkernel_Mins(int *gpuOA, int *gpuCA, int *gpulocals,
		int *gpucurrentupdate) {

	int id = blockIdx.x + 1;
	int threadId = gridDim.y * blockIdx.y + threadIdx.x;
	int thread = threadIdx.x;

	__shared__ int min;

	if (thread == 0) {
		min = gpulocals[id-1];
	}

	__syncthreads();

	int i = gpuOA[id - 1] + threadId;
	int j = gpuOA[id];
	if (i < j)
		atomicMin(&gpulocals[id - 1], gpucurrentupdate[gpuCA[i]]);
	__syncthreads();
	if (thread == 0) {
		atomicMin(&gpulocals[id - 1], min);
	}

}

// complete the following kernel...
__global__ void dkernel_Maxs(int *gpuOA, int *gpuCA, int *gpulocals,
		int *gpucurrentupdate) {

	int id = blockIdx.x + 1;
	int threadId = gridDim.y * blockIdx.y + threadIdx.x;
	int thread = threadIdx.x;

	__shared__ int max;

	if (thread == 0) {
		max = gpulocals[id-1];
	}

	__syncthreads();

	int i = gpuOA[id - 1] + threadId;
	int j = gpuOA[id];
	if (i < j)
		atomicMax(&gpulocals[id - 1], gpucurrentupdate[gpuCA[i]]);
	__syncthreads();
	if (thread == 0) {
		atomicMax(&gpulocals[id - 1], max);
	}

}

int main(int argc, char **argv) {

// variable declarations
	int m, n;
	int number;
	int numofquery;
	int op;
	struct timeval t1, t2;
	vector<double> kerneltime;

// File pointer declaration
	FILE *filePointer;

// File Opening for read
	char *filename = argv[1];
	filePointer = fopen(filename, "r");

// checking if file ptr is NULL
	if (filePointer == NULL) {
		printf("input.txt file failed to open.");
		return 0;
	}

	fscanf(filePointer, "%d", &n); // scaning the number of vertices
	fscanf(filePointer, "%d", &m); // scaning the number of edges

// D.S to store the input graph in COO format
	vector<edgepairs> COO(m);

// Reading from file and populate the COO
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < 2; j++) {
			if (fscanf(filePointer, "%d", &number) != 1)
				break;
			if (j % 2 == 0) {
				if (number >= 1 && number <= 10000)
					COO[i].x = number;
			} else {
				if (number >= 1 && number <= 10000)
					COO[i].y = number;
			}
		}
	}
// COO done...

// sort the COO
	sort(COO.begin(), COO.end(), compareTwoEdgePairs);
// sorting COO done..

// Converting the graph in COO format to CSR format..

// create the CSR

	int *OA = (int *) malloc((n + 1) * sizeof(int)); // Offsets Array
	for (int i = 0; i < n + 1; i++) {
		OA[i] = 0;
	}

	int *CA = (int *) malloc(m * sizeof(int)); // Coordinates Array
	OA[0] = 0;

// initialize the Coordinates Array
	for (int i = 0; i < m; i++) {
		if (COO[i].x >= 1 && COO[i].x <= 10000)
			CA[i] = COO[i].x - 1;
	}
// initialize the Offsets Array
	for (int i = 0; i < m; i++) {
		if (COO[i].y >= 1 && COO[i].y <= 10000)
			OA[COO[i].y]++; // store the frequency..
	}
	for (int i = 0; i < n; i++) {
		OA[i + 1] += OA[i]; // do cumulative sum..
	}

// Converting the graph to CSR done..

// copy initial local values to the array from the file
	int *initlocalvals = (int *) malloc(n * sizeof(int));
	;
	for (int i = 0; i < n; i++) {
		if (fscanf(filePointer, "%d", &number) != 1)
			break;

		initlocalvals[i] = number;
	}

// copying local vals end..

// get number of queries from the file
	fscanf(filePointer, "%d", &numofquery);

// copy OA,CA and initlocalvals to the GPU Memory
	int *gpuOA, *gpuCA, *gpulocals;
	hipMalloc(&gpuOA, sizeof(int) * (1 + n));
	hipMalloc(&gpuCA, sizeof(int) * m);
	hipMalloc(&gpulocals, sizeof(int) * n);
	hipMemcpy(gpuOA, OA, sizeof(int) * (1 + n), hipMemcpyHostToDevice);
	hipMemcpy(gpuCA, CA, sizeof(int) * m, hipMemcpyHostToDevice);
	hipMemcpy(gpulocals, initlocalvals, sizeof(int) * n,
			hipMemcpyHostToDevice);

	int *currentupdate = (int *) malloc(n * sizeof(int)); // array to store the updates that are pushed by each
														  // vertex to there neighbors
	int *gpucurrentupdate; // same as above but on GPU
	hipMalloc(&gpucurrentupdate, sizeof(int) * n);
	int *results = (int *) malloc(n * sizeof(int)); // storing the results from GPU to CPU for the enumerate query

// open the output.txt to write the query results
	char *fname = argv[2];
	FILE *fptr;
	fptr = fopen(fname, "w");

	for (int i = 0; i < numofquery; i++) {

		// read the operator
		fscanf(filePointer, "%d", &op);

		if (op != 3) { // if operator is other then enumerate (i.e. +,min,max)

			// read the current updates in the array
			for (int j = 0; j < n; j++) {
				if (fscanf(filePointer, "%d", &number) != 1)
					break;
				currentupdate[j] = number;
			}

			// copy current updates to gpu
			hipMemcpy(gpucurrentupdate, currentupdate, sizeof(int) * n,
					hipMemcpyHostToDevice);
			// kernel launches
			dim3 block_size(n, 100);
			if (op == 0) {
				gettimeofday(&t1, 0);

				dkernel_Adds<<<block_size, 1024>>>(gpuOA, gpuCA, gpulocals,
						gpucurrentupdate);
				hipDeviceSynchronize();
				gettimeofday(&t2, 0);
			}
			if (op == 1) {
				gettimeofday(&t1, 0);
				dkernel_Mins<<<block_size, 1024>>>(gpuOA, gpuCA, gpulocals,
						gpucurrentupdate);
				hipDeviceSynchronize();
				gettimeofday(&t2, 0);
			}
			if (op == 2) {
				gettimeofday(&t1, 0);
				dkernel_Maxs<<<block_size, 1024>>>(gpuOA, gpuCA, gpulocals,
						gpucurrentupdate);
				hipDeviceSynchronize();
				gettimeofday(&t2, 0);
			}

			double time = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec
					- t1.tv_usec) / 1000.0; // Time taken by kernel in seconds
			kerneltime.push_back(time);

			printf("Time taken by kernel to execute is: %.6f ms\n", time);
		}

		else { // if operator is enumnerate then store the results to file
			   // print local values of each vertices.
			hipMemcpy(results, gpulocals, n * sizeof(int),
					hipMemcpyDeviceToHost); // get each locals from GPU
			for (int j = 0; j < n; j++) {
				fprintf(fptr, "%d ", results[j]);
			}
			fprintf(fptr, "\n");
		}
	}

	int nall = kerneltime.size();
	double sumtime = 0;
	for (int i = 0; i < nall; i++) {
		sumtime += kerneltime[i];
	}
// print the time taken by all the kernels of the current test-case
	cout << "\ntotal time taken by the current test-case is " << sumtime
			<< " ms\n";

	fclose(fptr);
	fclose(filePointer);

	return 0;
}
