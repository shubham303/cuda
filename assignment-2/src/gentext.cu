
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>

using namespace std;

int main() {
	FILE *ptr = fopen("input.txt", "w");
	srand(time(NULL));
	int numberOfVertices = 10000, numberOfQueries = 40, numberOfAddQ = 10,
			numberOfMiinQ = 10, numberOfMaxQ = 10, numberOfPrintQ = 10;


	scanf("%d", &numberOfVertices);

	scanf("%d", &numberOfQueries);

	scanf("%d", &numberOfAddQ);

	scanf("%d", &numberOfMiinQ);

	scanf("%d", &numberOfMaxQ);

	int *edge = (int*) malloc(
			(numberOfVertices) * (numberOfVertices) * sizeof(int));

	memset(edge, 0, numberOfVertices * numberOfVertices * sizeof(int));
	int numberOfEdges = numberOfVertices * (numberOfVertices - 1) / 2;

	fprintf(ptr, "%d %d\n", numberOfVertices, numberOfEdges);

	for (long long i = 1; i <= numberOfEdges; i++) {
		int e1 = rand() % numberOfVertices;
		int e2 = rand() % numberOfVertices;
		while (edge[e1 * numberOfVertices + e2] == 1) {
			e1 = rand() % numberOfVertices;
			e2 = rand() % numberOfVertices;

		}
		fprintf(ptr, "%d %d\n", e1 + 1, e2 + 1);
		edge[e1 * numberOfVertices + e2] = 1;

	}

	for (int i = 0; i < numberOfVertices; i++) {
		int val = rand() % 1000;
		fprintf(ptr, "%d ", val);
	}
	fprintf(ptr, "\n");

	fprintf(ptr, "%d\n", numberOfQueries+1);

	while (numberOfQueries) {

		int q = rand() % 3;

		if (q == 1 && numberOfMiinQ) {
			fprintf(ptr, "1 ");
			for (int i = 0; i < numberOfVertices; i++) {
				int m = 0;
				while (m == 0)
					m = rand() % 1000;
				int val = rand() % m;
				fprintf(ptr, "%d ", val);
			}
			fprintf(ptr, "\n");
			numberOfMiinQ--;
			numberOfQueries--;
		}

		if (q == 2 && numberOfMaxQ) {
			fprintf(ptr, "2 ");
			for (int i = 0; i < numberOfVertices; i++) {
				int m = 0;
				while (m == 0)
					m = rand() % 1000;
				int val = rand() % m;
				fprintf(ptr, "%d ", val);
			}
			fprintf(ptr, "\n");
			numberOfMaxQ--;
			numberOfQueries--;
		}

		if (q == 0 && numberOfAddQ) {
			fprintf(ptr, "0 ");
			for (int i = 0; i < numberOfVertices; i++) {
				int m = 0;
				while (m == 0)
					m = rand() % 1000;
				int val = rand() % m;
				fprintf(ptr, "%d ", val);
			}
			fprintf(ptr, "\n");
			numberOfAddQ--;
			numberOfQueries--;
		}

	}

	fprintf(ptr, "3\n");
	numberOfPrintQ--;
	printf("\nCheck input.txt file\n");
	return 0;
}

