#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
using namespace std;

struct edgepairs {
	int x;
	int y;
};

int main(int argc, char **argv) {

	int m, n;
	int number;
	int numofquery;
	int op;
	vector<double> kerneltime;

	// File pointer declaration
	FILE *filePointer;

	// File Opening for read
	char *filename = argv[1];
	filePointer = fopen(filename, "r");

	// checking if file ptr is NULL
	if (filePointer == NULL) {
		printf("input.txt file failed to open.");
		return 0;
	}

	fscanf(filePointer, "%d", &n); // scaning the number of vertices
	fscanf(filePointer, "%d", &m); // scaning the number of edges

	// D.S to store the input graph in COO format
	vector<edgepairs> COO(m);

	// Reading from file and populate the COO
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < 2; j++) {
			if (fscanf(filePointer, "%d", &number) != 1)
				break;
			if (j % 2 == 0) {
				if (number >= 1 && number <= 10000)
					COO[i].x = number;
			} else {
				if (number >= 1 && number <= 10000)
					COO[i].y = number;
			}
		}
	}
	// COO done...

	int *initlocalvals = (int *) malloc(n * sizeof(int));
	;
	for (int i = 0; i < n; i++) {
		if (fscanf(filePointer, "%d", &number) != 1)
			break;

		initlocalvals[i] = number;
	}

	int *currentupdate = (int *) malloc(n * sizeof(int));

	char *fname = argv[2];
	FILE *fptr;
	fptr = fopen(fname, "w");
	fscanf(filePointer, "%d", &numofquery);
	for (int i = 0; i < numofquery; i++) {

		// read the operator
		fscanf(filePointer, "%d", &op);

		if (op != 3) { // if operator is other then enumerate (i.e. +,min,max)

			// read the current updates in the array
			for (int j = 0; j < n; j++) {
				if (fscanf(filePointer, "%d", &number) != 1)
					break;
				currentupdate[j] = number;
			}

			if (op == 0) {
				for (int i = 0; i < m; i++) {
					int x = COO[i].x;
					int y = COO[i].y;

					initlocalvals[y - 1] += currentupdate[x - 1];

				}
			}
			if (op == 1) {
				for (int i = 0; i < m; i++) {
					int x = COO[i].x;
					int y = COO[i].y;

					initlocalvals[y - 1] = min(initlocalvals[y - 1],
							currentupdate[x - 1]);

				}
			}
			if (op == 2) {
				int x = COO[i].x;
				int y = COO[i].y;

				initlocalvals[y - 1] = max(initlocalvals[y - 1],
						currentupdate[x - 1]);
			}
		}

		else { // if operator is enumnerate then store the results to file
			   // print local values of each vertices.

			for (int j = 0; j < n; j++) {
				fprintf(fptr, "%d ", initlocalvals[j]);
			}
			fprintf(fptr, "\n");
		}
	}

}
