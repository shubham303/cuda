/*
 * memory locality effect of GPU computation efficiency
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define N 1000000
#define BLOCKSIZE 1024

struct nodeAOS {
  int a;
  double b;

  char c;
} * allNodesAOS;

struct nodeSOA {
  int *a;
  double *b;
  char *c;
} soaNode;

__global__ void aosKernel(struct nodeAOS *allnodes) {
  unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
  allnodes[id].a = id;
  allnodes[id].b = id * 1.1;
  allnodes[id].c = 'c';
}

__global__ void soaKernel(int *a, double *b, char *c) {
  unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
  a[id] = id;
  b[id] = 1.1;
  c[id] = 'c';
}

double rtclock() {
  struct timezone Tzp;
  struct timeval Tp;
  int stat;
  stat = gettimeofday(&Tp, &Tzp);
  if (stat != 0)
    printf("Error return from gettimeofday: %d", stat);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime) {
  printf("%s%3f seconds\n", str, endtime - starttime);
}

int main() {
  hipMalloc(&allNodesAOS, N * sizeof(struct nodeAOS));
  // cudaMalloc(&soaNode, N * (sizeof(int) + sizeof(double) + sizeof(char)));

  hipMalloc(&soaNode.a, N * sizeof(int));
  hipMalloc(&soaNode.b, N * sizeof(double));
  hipMalloc(&soaNode.c, N * sizeof(char));

  int blocks = ceil((float)N / BLOCKSIZE);

  double startTime = rtclock();
  aosKernel<<<blocks, BLOCKSIZE>>>(allNodesAOS);
  hipDeviceSynchronize();
  double endTime = rtclock();
  printtime("AOS time", startTime, endTime);

  startTime = rtclock();
  soaKernel<<<blocks, BLOCKSIZE>>>(soaNode.a, soaNode.b, soaNode.c);
  hipDeviceSynchronize();
  endTime = rtclock();
  printtime("SOA time", startTime, endTime);
}
