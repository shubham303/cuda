// cuda program to initialize array elements.

#include <stdio.h> 
#include <hip/hip_runtime.h> 
#define N 8000

__global__ void arrayIntialization(int *a) {
    unsigned int tid=blockIdx.x*blockDim.x+threadIdx.x;
    if ( tid < N)
        a[tid] = 0;
}

__global__ void initializeArrayToNumber(int *a){
    unsigned int tid=(blockIdx.x*blockDim.x)+threadIdx.x;
    if ( tid < N)
        a[tid]+=tid;
   
}
int main() {
    int a[N], *da; 
    int i; 
    hipMalloc( &da, N * sizeof(int)); 
    arrayIntialization<<<8,1024>>>(da);
    hipMemcpy(a, da, N * sizeof(int), hipMemcpyDeviceToHost); 
   
    //for (i = 0; i < N; i++)
//        printf("%d ", a[i]); 

    printf("\n \n");
    initializeArrayToNumber<<<8,1024>>>(da);
    hipDeviceSynchronize();
    hipMemcpy(a, da, N*sizeof(int), hipMemcpyDeviceToHost);


    for (i = 0; i < N; i++)
        printf("%d ", a[i]); 

                    return 0; 

}