// program to print n Numbers


#include<stdio.h>
#include<hip/hip_runtime.h>

#define N 100

__global__ void print(){
    if(threadIdx.x<N)
        printf("%d ",threadIdx.x*threadIdx.x);

}
int main(){
    int i;

    printf("numbers printing by CPU \n ");

    for( i=0;i<N ;i++){
        printf("%d ",i*i);
    }
    printf("\n numbers printing by GPU\n");

    print<<<1,N>>>();
    hipDeviceSynchronize();


    return 0;

}