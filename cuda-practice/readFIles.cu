/*
Read a sequence of integers from a file.
● Square each number.
● Read another sequence of integers from
another file.
● Cube each number.
● Sum the two sequences element-wise, store in
the third sequence.
● Print the computed sequence.
*/

#include<stdio.h>
#include<hip/hip_runtime.h>

#define N 2
#define M 32


__global__ void square(int* da, int length){
    unsigned id=blockIdx.x*blockDim.x+threadIdx.x;
    if(id<length){
        da[id]=da[id]*da[id];
       
    }
}

__global__  void cube(int *da, int length){
    unsigned id=blockIdx.x*blockDim.x+threadIdx.x;
    if(id<length){
        da[id]=da[id]*da[id]*da[id];
    }
}

__global__ void add(int* a, int *b, int length){
    unsigned id=blockIdx.x*blockDim.x+threadIdx.x;
    if(id<length){
        a[id]+=b[id];
    }
}

int main(){
    FILE *f;
    int array[1024];
    int *a,*b;

   int length=0;
    int data;
    char aux;
    
    f=fopen("numbers1.txt","r");

    while(EOF!=fscanf(f,"%d%c",&data,&aux)){
        array[length]=data;
        length++;
    }


    
    hipMalloc(&a, length*sizeof(int));
    hipMemcpy(a,array,length*sizeof(int), hipMemcpyHostToDevice);
    square<<<N,M>>>(a, length);

    length=0; 
    f=fopen("numbers2.txt","r");
    while(EOF!=fscanf(f,"%d%c",&data,&aux)){
        array[length]=data;
        length++;
    }
    hipMalloc(&b, length*sizeof(int));
    hipMemcpy(b,array, length*sizeof(int), hipMemcpyHostToDevice);
    cube<<<N,M>>>(b,length);
    add<<<N,M>>>(a,b,length);

    hipMemcpy(array, a, length*sizeof(int),hipMemcpyDeviceToHost);

    for(int i=0;i<length;i++){
        printf("%d ",array[i]);
    }
    
}