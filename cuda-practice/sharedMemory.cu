#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void dkernel(){
    __shared__ int s;
    if(threadIdx.x==0) s=0;

    if(threadIdx.x==1)s+=1;
   // __syncthreads();

    if(threadIdx.x==122) s+=2;
    __syncthreads();
    if(threadIdx.x==0) printf("%d",s);
}

int main(){
    int i;
    for(i=0;i<100;i++){
        dkernel<<<2,1024>>>();
        hipDeviceSynchronize();
    }
}