#include "hip/hip_runtime.h"
__global__ void init(int ** matrix, int N, int M){
    unsigned id=threadIdx.x * blockDim.y + threadIdx.y;
    if (id< N*M){
        matrix[id]=id;
    }
}